﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <cstdint>

#include "cuda_helper.cuh"
#include "ConcurrentStreamParams.cuh"

#define NUM_GPUS 1

int main()
{
    hipDeviceProp_t cudaProperties;

    for (int i = 0; i < NUM_GPUS; i++) {
        checkCuda(hipSetDevice(i), "Set Device");
        checkCuda(hipGetDeviceProperties(&cudaProperties, i), "Get Device Properties");
        std::cout << cudaProperties.name << std::endl;
    }

    return 0;
}