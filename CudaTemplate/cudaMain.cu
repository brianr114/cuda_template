﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <cstdint>

#include "cuda_helper.cuh"

#define NUM_GPUS 1

int main()
{
    int deviceId{ 0 };
    hipDeviceProp_t cudaProperties;

    for (int i = 0; i < NUM_GPUS; i++) {
        checkCuda(hipSetDevice(i), "Set Device");
        checkCuda(hipGetDeviceProperties(&cudaProperties, i), "Get Device Properties");
        std::cout << cudaProperties.name << std::endl;
    }

    return 0;
}