﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result, const char *error_step)
{
    if (result != hipSuccess) {
        printf("CUDA Step: %s\nCUDA Runtime Error: %s\n", error_step, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

int main()
{
    int deviceId;
    hipDeviceProp_t cudaProperties;

    checkCuda(hipGetDevice(&deviceId), "Get Device ID");
    checkCuda(hipGetDeviceProperties(&cudaProperties, deviceId), "Get Device Properties");
    
    return 0;
}