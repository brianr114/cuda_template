#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result, const char *error_step)
{
    if (result != hipSuccess) {
        printf("CUDA Step: %s\nCUDA Runtime Error: %s\n", error_step, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

void calc_concurrent_stream_addr(unsigned long long N, unsigned long long num_streams, unsigned long long iterator, unsigned long long data_size, unsigned long long& lower, unsigned long long& width);

int main()
{
    int deviceId;
    hipDeviceProp_t cudaProperties;

    checkCuda(hipGetDevice(&deviceId), "Get Device ID");
    checkCuda(hipGetDeviceProperties(&cudaProperties, deviceId), "Get Device Properties");

    return 0;
}

void calc_concurrent_stream_addr(unsigned long long N, unsigned long long num_streams, unsigned long long iterator, unsigned long long data_size, unsigned long long& lower, unsigned long long& width)
{
    unsigned long long chunk_size = (N + num_streams - 1) / num_streams;
    lower = chunk_size * iterator; 
    unsigned long long upper = min(lower + chunk_size, N);
    width = (upper - lower) * data_size;
}