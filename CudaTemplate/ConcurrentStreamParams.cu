#include "hip/hip_runtime.h"
#include <algorithm>
#include "ConcurrentStreamParams.cuh"

ConcurrentStreamParams::ConcurrentStreamParams(uint64_t num_elements, uint64_t num_gpus, uint64_t num_streams)
{
	ConcurrentStreamParams::num_elements = num_elements;
	ConcurrentStreamParams::num_gpus = num_gpus;
	ConcurrentStreamParams::streams_per_gpu = num_streams;
	ConcurrentStreamParams::stream_chunk = sdiv(sdiv(num_elements, num_gpus), num_streams);
	ConcurrentStreamParams::gpu_chunk = ConcurrentStreamParams::stream_chunk * num_streams;
}

ConcurrentStreamParams::~ConcurrentStreamParams()
{
}

buffer_chunk ConcurrentStreamParams::calc_gpu_chunk(uint64_t gpu)
{
	buffer_chunk params;

	params.lower = gpu_chunk * gpu;
	params.upper = std::min(params.lower + gpu_chunk, num_elements);
	params.width = params.upper - params.lower;

	return params;
}

buffer_chunk ConcurrentStreamParams::calc_stream_chunk(uint64_t gpu, uint64_t stream)
{
	buffer_chunk params;

	params.stream_offset = stream_chunk * stream;
	params.lower = gpu_chunk * gpu + params.stream_offset;
	params.upper = std::min(params.lower + stream_chunk, num_elements);
	params.width = params.upper - params.lower;
	
	return params;
}
